
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>  // for rand()
#include <ctime>    // for seeding rand()
#include <cmath>    // for abs

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

/* TODO: Experiment with the following values in turn to observe the change in duration difference between kernels */
const size_t N = 8ULL*1024ULL*1024ULL;  // 8M
//const size_t N = 256*640; // 163840
//const size_t N = 32ULL*1024ULL*1024ULL; // 32M

const int BLOCK_SIZE = 256;  // CUDA maximum is 1024

/**
 * @brief CUDA kernel for atomic reduction.
 * 
 * Each thread adds an element from the input array `gdata` to a shared output
 * using atomic operations to avoid race conditions.
 * 
 * @param gdata Pointer to the input data (in device memory).
 * @param out   Pointer to the output sum (in device memory).
 * 
 * Details:
 * - This kernel uses atomic operations to ensure that multiple threads can safely
 *   update the shared `out` variable simultaneously.
 */
__global__ void atomic_red(const int *gdata, int *out) { 
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        /* TODO: Use atomicAdd to safely add the value from gdata to the output sum */
        atomicAdd(out, gdata[idx]);
    }
}

/**
 * @brief CUDA kernel for parallel reduction using atomic operations.
 * 
 * This kernel performs reduction similar to the previous one, but uses atomicAdd
 * to accumulate the final result.
 * 
 * @param gdata Pointer to the input data (in device memory).
 * @param out   Pointer to the output sum (in device memory).
 */
__global__ void reduce_a(int *gdata, int *out) {
    /* TODO: Declare shared memory for this block */
    __shared__ int temp[BLOCK_SIZE];

    /* TODO: Initialize thread-specific local sum in shared memory */
    temp[threadIdx.x] = 0;

    /* TODO: Calculate global thread index */
    int gidx = threadIdx.x + blockDim.x * blockIdx.x;

    /* TODO: Load data in chunks using grid-stride loop, accumulating the sum */
    int stride = blockDim.x * gridDim.x;
    for (int i = gidx; i < N; i += stride) {
        temp[threadIdx.x] += gdata[i];
    }

    /* TODO: Perform parallel reduction to sum the elements in shared memory */
    for (int j = 2; j < BLOCK_SIZE; j *= 2) {
        if (threadIdx.x % j == 0)
            temp[threadIdx.x] += temp[threadIdx.x + (j / 2)];
        __syncthreads();
    }

    /* TODO: Use atomicAdd to accumulate the final result safely */
    if (threadIdx.x == 0)
        atomicAdd(out, temp[0]);
}

/**
 * @brief CUDA kernel for warp-shuffle based parallel reduction.
 * 
 * This kernel uses warp-shuffle instructions to perform a reduction within a warp.
 * The result of each warp is stored in shared memory, and then a final warp reduces
 * the results from all warps.
 * 
 * @param gdata Pointer to the input data (in device memory).
 * @param out   Pointer to the output sum (in device memory).
 */
__global__ void reduce_ws(int *gdata, int *out) {
    /* TODO: Declare shared memory to hold the results of each warp(32 threads) */
    __shared__ int temp[8];
    int tid = threadIdx.x;
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int val = 0;
    unsigned mask = 0xFFFFFFFFU;
    int lane = threadIdx.x % warpSize;  // Lane within a warp
    int warpID = threadIdx.x / warpSize;  // Warp ID

    /* TODO: Load data in grid-stride loop, accumulating in val */
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride)
        val += gdata[i];
    __syncthreads();

    /* TODO: Perform warp-level reduction using __shfl_down_sync (within each warp) */
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(mask, val, offset);
    
    /* TODO: Write the warp's result to shared memory */
    if (lane == 0)
        temp[warpID] = val;
    __syncthreads();

    /* TODO: If warp 0, perform final reduction on the values from each warp */
    if (warpID == 0) {
        int total_sum = temp[lane];
        total_sum += __shfl_down_sync(mask, total_sum, 4);
        total_sum += __shfl_down_sync(mask, total_sum, 2);
        total_sum += __shfl_down_sync(mask, total_sum, 1);
        /* TODO: Use atomicAdd to safely add the final result to the global sum */
        if (tid == 0)
            atomicAdd(out, total_sum);
    }
}


int main() {
    int *h_A, *h_sum, *d_A, *d_sum;
    h_A = new int[N];  // Allocate space for data in host memory
    h_sum = new int;

    
    srand(time(0));
    int cpu_sum = 0;
    for (size_t i = 0; i < N; i++) {
        h_A[i] = rand() % 100;  // Random int between 0 and 99
        cpu_sum += h_A[i];  // Calculate CPU sum for validation
    }

    hipMalloc(&d_A, N * sizeof(int));  
    hipMalloc(&d_sum, sizeof(int));  
    cudaCheckErrors("hipMalloc failure");

    hipMemcpy(d_A, h_A, N * sizeof(int), hipMemcpyHostToDevice);
    cudaCheckErrors("hipMemcpy H2D failure");

    hipMemset(d_sum, 0, sizeof(int));
    cudaCheckErrors("hipMemset failure");

    atomic_red<<<(N + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_A, d_sum);
    cudaCheckErrors("atomic reduction kernel launch failure");

    hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("atomic reduction kernel execution failure or hipMemcpy H2D failure");

    if (abs(*h_sum - cpu_sum) > 0) {
        printf("atomic sum reduction incorrect! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);
        return -1;
    }
    printf("atomic sum reduction correct! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);

    const int blocks = 640;
    hipMemset(d_sum, 0, sizeof(int));
    cudaCheckErrors("hipMemset failure");

    reduce_a<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
    cudaCheckErrors("reduction w/atomic kernel launch failure");

    hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("reduction w/atomic kernel execution failure or hipMemcpy H2D failure");

    if (abs(*h_sum - cpu_sum) > 0) {
        printf("reduction w/atomic sum incorrect! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);
    }
    printf("reduction w/atomic sum correct! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);

    hipMemset(d_sum, 0, sizeof(int));
    cudaCheckErrors("hipMemset failure");

    reduce_ws<<<blocks, BLOCK_SIZE>>>(d_A, d_sum);
    cudaCheckErrors("reduction warp shuffle kernel launch failure");

    hipMemcpy(h_sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
    cudaCheckErrors("reduction warp shuffle kernel execution failure or hipMemcpy H2D failure");

    if (abs(*h_sum - cpu_sum) > 0) {
        printf("reduction warp shuffle sum incorrect! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);
        return -1;
    }
    printf("reduction warp shuffle sum correct! CPU: %d, GPU: %d\n", cpu_sum, *h_sum);

    delete[] h_A;
    delete h_sum;
    hipFree(d_A);
    hipFree(d_sum);

    return 0;
}
