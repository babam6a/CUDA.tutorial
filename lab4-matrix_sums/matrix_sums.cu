
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>  // for rand()
#include <ctime>    // for seeding rand()
#include <cmath>    // for abs

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

const size_t DSIZE = 16384;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024

/**
 * @brief CUDA kernel for calculating row sums of a matrix.
 * 
 * Each thread calculates the sum of one row in the matrix and stores the result
 * in the corresponding index of the output vector `sums`.
 * 
 * @param A     Pointer to the input matrix (1D array representing a 2D matrix) in device memory.
 * @param sums  Pointer to the output vector (sums of each row) in device memory.
 * @param ds    The dimension size of the matrix (number of rows and columns).
 *
 * Details:
 * - Each thread is assigned to a single row of the matrix.
 * - The thread iterates through all columns of its assigned row, summing up the elements.
 */
__global__ void row_sums(const float *A, float *sums, size_t ds){

  /* TODO: Calculate the global thread index to map threads to matrix rows. */
  int idx = threadIdx.x + blockDim.x * blockIdx.x; 

  if (idx < ds) {
    float sum = 0.0f;
    /* TODO: Iterate through the columns of the matrix, accumulating the sum for the row. */
    for (int i = 0; i < ds; i++) {
      sum += A[idx * ds + i];
    }
    sums[idx] = sum;
  }
}

/**
 * @brief CUDA kernel for calculating column sums of a matrix.
 * 
 * Each thread calculates the sum of one column in the matrix and stores the result
 * in the corresponding index of the output vector `sums`.
 * 
 * @param A     Pointer to the input matrix (1D array representing a 2D matrix) in device memory.
 * @param sums  Pointer to the output vector (sums of each column) in device memory.
 * @param ds    The dimension size of the matrix (number of rows and columns).
 *
 * Details:
 * - Each thread is assigned to a single column of the matrix.
 * - The thread iterates through all rows of its assigned column, summing up the elements.
 */
__global__ void column_sums(const float *A, float *sums, size_t ds){

  /* TODO: Calculate the global thread index to map threads to matrix columns. */
  int idx = threadIdx.x + blockDim.x * blockIdx.x; 

  if (idx < ds) {
    float sum = 0.0f;
    /* TODO: Iterate through the rows of the matrix, accumulating the sum for the column. */
    for (int i = 0; i < ds; i++) {
      sum += A[i * ds + idx];
    }
    sums[idx] = sum;
  }
}

// CPU implementation for row sums
void cpu_row_sums(const float *A, float *sums, size_t ds) {
    for (size_t row = 0; row < ds; row++) {
        float sum = 0.0f;
        for (size_t col = 0; col < ds; col++) {
            sum += A[row * ds + col];  // Accumulate sum across columns in a row
        }
        sums[row] = sum;
    }
}

// CPU implementation for column sums
void cpu_column_sums(const float *A, float *sums, size_t ds) {
    for (size_t col = 0; col < ds; col++) {
        float sum = 0.0f;
        for (size_t row = 0; row < ds; row++) {
            sum += A[col + ds * row];  // Accumulate sum across rows in a column
        }
        sums[col] = sum;
    }
}

bool validate(const float *gpu_results, const float *cpu_results, size_t sz){
  for (size_t i = 0; i < sz; i++) {
    if (abs(gpu_results[i] - cpu_results[i]) > 1e-2) {  // Allow small floating-point tolerance
      printf("Results mismatch at index %lu, GPU result: %f, CPU result: %f\n", i, gpu_results[i], cpu_results[i]);
      return false;
    }
  }
  return true;
}

int main() {
  float *h_A, *h_sums, *h_sums_CPU, *d_A, *d_sums;
  h_A = new float[DSIZE * DSIZE];  // allocate space for matrix in host memory
  h_sums = new float[DSIZE]();     // allocate space for GPU results in host memory
  h_sums_CPU = new float[DSIZE](); // allocate space for CPU validation sums

  // Seed the random number generator
  srand(time(0));  // Use current time to seed the random number generator

  // Initialize matrix in host memory with random values
  for (int i = 0; i < DSIZE * DSIZE; i++) {
    h_A[i] = static_cast<float>(rand()) / RAND_MAX;  // Random float between 0 and 1
  }

  hipMalloc(&d_A, DSIZE * DSIZE * sizeof(float));  // allocate device space for matrix A
  hipMalloc(&d_sums, DSIZE * sizeof(float));       // allocate device space for sums vector
  cudaCheckErrors("hipMalloc failure"); // error checking

  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE * DSIZE * sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");

  //cuda processing sequence step 1 is complete
  row_sums<<<(DSIZE + block_size - 1) / block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // copy row sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("kernel execution failure or hipMemcpy D2H failure");

  // CPU row sums calculation for validation
  cpu_row_sums(h_A, h_sums_CPU, DSIZE);

  if (!validate(h_sums, h_sums_CPU, DSIZE)) return -1;
  printf("Row sums correct!\n");

  hipMemset(d_sums, 0, DSIZE * sizeof(float));  // reset sums on device

  column_sums<<<(DSIZE + block_size - 1) / block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");

  // copy column sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE * sizeof(float), hipMemcpyDeviceToHost);
  cudaCheckErrors("kernel execution failure or hipMemcpy D2H failure");

  // CPU column sums calculation for validation
  cpu_column_sums(h_A, h_sums_CPU, DSIZE);

  if (!validate(h_sums, h_sums_CPU, DSIZE)) return -1;
  printf("Column sums correct!\n");

  // Cleanup
  delete[] h_A;
  delete[] h_sums;
  delete[] h_sums_CPU;
  hipFree(d_A);
  hipFree(d_sums);

  return 0;
}
