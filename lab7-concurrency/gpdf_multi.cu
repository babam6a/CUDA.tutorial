
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>

// modifiable
typedef float ft;
const int chunks = 64;
const size_t ds = 1024*1024*chunks;
const int count = 22;
const int num_gpus = 2;

// not modifiable
const float sqrt_2PIf = 2.5066282747946493232942230134974f;
const double sqrt_2PI = 2.5066282747946493232942230134974;

/**
 * @brief Compute the Gaussian PDF for a given value.
 * 
 * This function calculates the Gaussian probability density function (PDF)
 * for a given value and standard deviation.
 * 
 * @param val   The value for which the PDF is computed.
 * @param sigma The standard deviation for the Gaussian distribution.
 * @return float The computed Gaussian PDF value.
 */
__device__ float gpdf(float val, float sigma) {
    return expf(-0.5f * val * val) / (sigma * sqrt_2PIf);
}

/**
 * @brief Compute the Gaussian PDF for a given double value.
 * 
 * Similar to the float version, but works with double precision.
 * 
 * @param val   The value for which the PDF is computed.
 * @param sigma The standard deviation for the Gaussian distribution.
 * @return double The computed Gaussian PDF value.
 */
__device__ double gpdf(double val, double sigma) {
    return exp(-0.5 * val * val) / (sigma * sqrt_2PI);
}

/**
 * @brief CUDA kernel to compute the average Gaussian PDF over a window.
 * 
 * This kernel computes the average Gaussian PDF value over a window of values
 * around each point. It processes the data in parallel using CUDA threads.
 * 
 * @param x     Pointer to the input data array.
 * @param y     Pointer to the output data array.
 * @param mean  The mean of the Gaussian distribution.
 * @param sigma The standard deviation of the Gaussian distribution.
 * @param n     The number of data points.
 */
__global__ void gaussian_pdf(const ft * __restrict__ x, ft * __restrict__ y, const ft mean, const ft sigma, const int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        ft in = x[idx] - (count / 2) * 0.01f;
        ft out = 0;
        for (int i = 0; i < count; i++) {
            ft temp = (in - mean) / sigma;
            out += gpdf(temp, sigma);
            in += 0.01f;
        }
        y[idx] = out / count;
    }
}

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// host-based timing
#define USECPSEC 1000000ULL

/**
 * @brief Measure elapsed time in microseconds.
 * 
 * This function calculates the elapsed time in microseconds since the provided
 * start time.
 * 
 * @param start  The start time in microseconds.
 * @return unsigned long long The elapsed time in microseconds.
 */
unsigned long long dtime_usec(unsigned long long start) {
    timeval tv;
    gettimeofday(&tv, 0);
    return ((tv.tv_sec * USECPSEC) + tv.tv_usec) - start;
}

int main() {
    // Check Device number
    int count;
    hipGetDeviceCount(&count);
    if (count < num_gpus) {
        fprintf(stderr, "You should run this code with %d or more Devices! Current: %d\n", num_gpus, count);
        return -1;
    }

    ft *h_x, *d_x[num_gpus], *d_y[num_gpus];

    h_x = (ft *)malloc(ds * sizeof(ft));
    for (int i = 0; i < num_gpus; i++) {
        /* 
        * TODO: Allocate device memory using cudaMalloc for each GPU.
        * Allocate input (d_x) and output (d_y) memory on both GPUs.
        */
        hipSetDevice(i);
        hipMalloc(&d_x[i], ds * sizeof(ft));
        hipMalloc(&d_y[i], ds * sizeof(ft));
    }
    cudaCheckErrors("allocation error");

    
    for (int i = 0; i < num_gpus; i++) {
        for (size_t j = 0; j < ds; j++) {
            h_x[j] = rand() / (ft)RAND_MAX;
        }
       /* 
        * TODO: Perform memory copies to transfer input data from the host (h_x) to each GPU.
        */
        hipMemcpy(d_x[i], h_x, ds * sizeof(ft), hipMemcpyHostToDevice);
    }
    cudaCheckErrors("copy error");
    unsigned long long et1 = dtime_usec(0);

    /* 
     * TODO: Launch the gaussian_pdf kernel on each GPU in parallel.
     * This loop sets the current device and launches the kernel.
     */
    for (int i = 0; i < num_gpus; i++) {
        hipSetDevice(i);
        gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x[i], d_y[i], 0.0, 1.0, ds);
    }

    hipDeviceSynchronize();
    
    cudaCheckErrors("execution error");
    et1 = dtime_usec(et1);
    std::cout << "elapsed time: " << et1 / (float)USECPSEC << std::endl;

    return 0;
}
