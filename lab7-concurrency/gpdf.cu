
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include <sys/time.h>
#include <stdio.h>

// modifiable
typedef float ft;
const int chunks = 64;
const size_t ds = 1024*1024*chunks;
const int count = 22;
const int num_streams = 8;

// not modifiable
const float sqrt_2PIf = 2.5066282747946493232942230134974f;
const double sqrt_2PI = 2.5066282747946493232942230134974;

/**
 * @brief Compute the Gaussian PDF for a given value.
 * 
 * This function calculates the Gaussian probability density function (PDF)
 * for a given value and standard deviation.
 * 
 * @param val   The value for which the PDF is computed.
 * @param sigma The standard deviation for the Gaussian distribution.
 * @return float The computed Gaussian PDF value.
 */
__device__ float gpdf(float val, float sigma) {
    return expf(-0.5f * val * val) / (sigma * sqrt_2PIf);
}

/**
 * @brief Compute the Gaussian PDF for a given double value.
 * 
 * Similar to the float version, but works with double precision.
 * 
 * @param val   The value for which the PDF is computed.
 * @param sigma The standard deviation for the Gaussian distribution.
 * @return double The computed Gaussian PDF value.
 */
__device__ double gpdf(double val, double sigma) {
    return exp(-0.5 * val * val) / (sigma * sqrt_2PI);
}

/**
 * @brief CUDA kernel to compute the average Gaussian PDF over a window.
 * 
 * This kernel computes the average Gaussian PDF value over a window of values
 * around each point. It processes the data in parallel using CUDA threads.
 * 
 * @param x     Pointer to the input data array.
 * @param y     Pointer to the output data array.
 * @param mean  The mean of the Gaussian distribution.
 * @param sigma The standard deviation of the Gaussian distribution.
 * @param n     The number of data points.
 */
__global__ void gaussian_pdf(const ft * __restrict__ x, ft * __restrict__ y, const ft mean, const ft sigma, const int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        ft in = x[idx] - (count / 2) * 0.01f;
        ft out = 0;
        for (int i = 0; i < count; i++) {
            ft temp = (in - mean) / sigma;
            out += gpdf(temp, sigma);
            in += 0.01f;
        }
        y[idx] = out / count;
    }
}

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// host-based timing
#define USECPSEC 1000000ULL

/**
 * @brief Measure elapsed time in microseconds.
 * 
 * This function calculates the elapsed time in microseconds since the provided
 * start time.
 * 
 * @param start  The start time in microseconds.
 * @return unsigned long long The elapsed time in microseconds.
 */
unsigned long long dtime_usec(unsigned long long start) {
    timeval tv;
    gettimeofday(&tv, 0);
    return ((tv.tv_sec * USECPSEC) + tv.tv_usec) - start;
}

int main() {
    ft *h_x, *d_x, *h_y, *h_y1, *d_y;

    /* 
     * TODO: Allocate memory using malloc and device memory using cudaMalloc.
     * The memory is allocated for both input (h_x) and output (h_y, h_y1) arrays.
     */
    hipMalloc(&d_x, ds*sizeof(ft));
    hipMalloc(&d_y, ds*sizeof(ft));
    h_x = (ft *)malloc(ds*sizeof(ft));
    h_y = (ft *)malloc(ds*sizeof(ft));
    h_y1 = (ft *)malloc(ds*sizeof(ft));

    cudaCheckErrors("allocation error");

    gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds);

    for (size_t i = 0; i < ds; i++) {
        h_x[i] = rand() / (ft)RAND_MAX;
    }
    hipDeviceSynchronize();

    unsigned long long et1 = dtime_usec(0);

    hipMemcpy(d_x, h_x, ds * sizeof(ft), hipMemcpyHostToDevice);
    gaussian_pdf<<<(ds + 255) / 256, 256>>>(d_x, d_y, 0.0, 1.0, ds);
    hipMemcpy(h_y1, d_y, ds * sizeof(ft), hipMemcpyDeviceToHost);
    cudaCheckErrors("non-streams execution error");

    et1 = dtime_usec(et1);
    std::cout << "non-stream elapsed time: " << et1 / (float)USECPSEC << std::endl;

    return 0;
}
